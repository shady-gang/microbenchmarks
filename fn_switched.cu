#include "hip/hip_runtime.h"
#include <cstdint>

extern "C" {

__noinline__ __device__ int f(int x, int y) {
    return x + y;
}

__noinline__ __device__ int g(int x, int y) {
    return x * y;
}

__noinline__ __device__ int h(int x, int y) {
    return x - y;
}

__noinline__ __device__ int k(int x, int y) {
    if (y == 0)
        return x;
    return x / y;
}

__global__ void fn_ptrs(__device__ int32_t* a, __device__ int32_t* b, __device__ int32_t* c) {
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    switch (i % 4) {
    case 0: c[i] = f(a[i], b[i]) + 41; return;
    case 1: c[i] = g(a[i], b[i]) + 41; return;
    case 2: c[i] = h(a[i], b[i]) + 41; return;
    case 3: c[i] = k(a[i], b[i]) + 41; return;
    }
}

}