#include "hip/hip_runtime.h"
//#include <shady.h>
#include <stdint.h>

//using namespace vcc;

#define FN_PREFIX
#include "unions.h"

#define w 256

extern "C" {

/*void debug_printff(const char*, float) __asm__("shady::prim_op::debug_printf::i");
void debug_printfi(const char*, int) __asm__("shady::prim_op::debug_printf::i");*/

__device__
S patch(S s, int o) {
  if (s.tag) {
    s.f += (float) o;
    //debug_printff("f%f", s.f);
  } else {
    s.i += o;
    //debug_printfi("i%d", s.i);
  }
  return s;
}

__device__
unsigned int hash_fnv(char* str, unsigned int length) {
    const unsigned int fnv_prime = 0x811C9DC5;
    unsigned int hash = 0;
    unsigned int i = 0;

    for (i = 0; i < length; str++, i++)
    {
        hash *= fnv_prime;
        hash ^= (*str);
    }

    return hash;
}

__global__
void unions_kernel(__device__ S* a, __device__ unsigned int* b) {
    //int i = gl_GlobalInvocationID.x;
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    S arr[16];
    for (int j = 0; j < 16; j++) {
        arr[j] = patch(a[j], i);
    }
    b[i] = hash_fnv((char*) &arr[0], sizeof(arr));
}

}
