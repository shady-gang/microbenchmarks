#include "hip/hip_runtime.h"
#include <cstdint>

extern "C" {

__device__ int f(int x, int y) {
    return x + y;
}

__device__ int g(int x, int y) {
    return x * y;
}

__device__ int h(int x, int y) {
    return x - y;
}

__device__ int k(int x, int y) {
    if (y == 0)
        return x;
    return x / y;
}

__global__ void fn_ptrs(__device__ int32_t* a, __device__ int32_t* b, __device__ int32_t* c) {
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    int (*fns[4])(int, int) = { f, g, h, k };
    int (*fn)(int, int) = fns[i % 4];

    c[i] = fn(a[i], b[i]) + 41;
}

}