#include "hip/hip_runtime.h"
#include <cstdint>

extern "C" {

__device__ int fib(int n) {
    if (n <= 1)
        return 1;
    return fib(n - 1) + fib(n - 2);
}

struct Fib {
    int fib_stack_size = 0;
    int fib_stack[32];

    __device__ int fib_stack_unwind(int r) {
        if (fib_stack_size > 0) {
            int t = fib_stack[--fib_stack_size];
            switch (t) {
                case 0: {
                    int n = fib_stack[--fib_stack_size];
                    return fib_(r, n);
                }
                case 1: {
                    int r2 = fib_stack[--fib_stack_size];
                    return fib_stack_unwind(r + r2);
                }
            }
        }
        return r;
    }

    __device__ int fib_(int r, int n) {
        fib_stack[fib_stack_size++] = r;
        fib_stack[fib_stack_size++] = 1;
        return fib(n);
    }

    __device__ int fib(int n) {
        if (n <= 1)
            return fib_stack_unwind(1);
        fib_stack[fib_stack_size++] = n - 2;
        fib_stack[fib_stack_size++] = 0;
        return fib(n - 1);
    }

    /*__device__ int fib(int n) {
        int r;

        while (1) {
            //debug_printfi("n=%d\n", n);
            if (n <= 1) {
                //debug_printfi("unwind, old_r=%d\n", r);
                r = 1;
                bool unwind = true;
                while (unwind) {
                    //debug_printfi("unwind iter, fib_stack_size=%d\n", fib_stack_size);
                    if (fib_stack_size > 0) {
                        int t = fib_stack[--fib_stack_size];
                        switch (t) {
                            case 0: {
                                n = fib_stack[--fib_stack_size];
                                //debug_printfi("popped n=%d\n", n);
                                fib_stack[fib_stack_size++] = r;
                                fib_stack[fib_stack_size++] = 1;
                                unwind = false;
                                break;
                            }
                            case 1: {
                                int r2 = fib_stack[--fib_stack_size];
                                r = r + r2;
                                //debug_printfi("r=%d\n", r);
                                break;
                            }
                        }
                        continue;
                    }
                    //debug_printfi("return=%d\n", r);
                    return r;
                }
                continue;
            }
            fib_stack[fib_stack_size++] = n - 2;
            fib_stack[fib_stack_size++] = 0;
            n = n - 1;
        }
    }*/
};

__global__ void fib_main(__device__ int32_t* a) {
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    Fib f {};
    a[i] = f.fib(i % 16);
}

}