#include "hip/hip_runtime.h"
#include <cstdint>

extern "C" {

__device__ int fib(int n) {
    if (n <= 1)
        return 1;
    return fib(n - 1) + fib(n - 2);
}

__global__ void fib_main(__device__ int32_t* a) {
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    a[i] = fib(i % 16);
}

}