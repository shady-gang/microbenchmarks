#include "hip/hip_runtime.h"
#include <cstdint>

extern "C" {

__global__ void add_buffers(__device__ int32_t* a, __device__ int32_t * b , __device__ int32_t* c ) {
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    c[i] = a[i] + b[i];
}

}