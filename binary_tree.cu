#include "hip/hip_runtime.h"
#include <cstdint>

#define FN_PREFIX __device__
#include "binary_tree.h"

extern "C" {

using Node = TreeNode<int>;

__global__ void treesearch(__device__ int32_t* a, __device__ int32_t * b , __device__ Node* root ) {
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    b[i] = !!find((Node*) root, a[i]);
}

}
